#include <iostream>
#include "hip/hip_runtime.h"


#include <time.h>

__global__ void AddInts(int32_t* a, int32_t* b, int32_t count)
{
	int32_t id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count) {
		a[id] += b[id];
	}
}
int main( void ) {
	hipError_t err = hipSuccess;

	srand(time(NULL));
	const int16_t count = 100;
	int32_t* h_a = new int32_t[count];
	int32_t* h_b = new int32_t[count];
	for (int32_t i = 0; i < count; i++)
	{
		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000;
	}
	std::cout << "Prior to addition : " << std::endl;
	for (int32_t i = 0; i < 5; i++)
	{
		std::cout << h_a[i] << " " << h_b[i] << std::endl;
	}
	int32_t* d_a,* d_b;

	err = hipMalloc(&d_a, sizeof(int32_t) * count);
	err = hipMalloc(&d_b, sizeof(int32_t) * count);

	err = hipMemcpy(d_a, h_a, sizeof(int32_t) * count, hipMemcpyHostToDevice);
	err = hipMemcpy(d_b, h_b, sizeof(int32_t) * count, hipMemcpyHostToDevice);

	AddInts<<<count/256 + 1, 256>>>(d_a, d_b,count);	

	err = hipMemcpy(h_a, d_a,sizeof(int32_t) * count, hipMemcpyDeviceToHost);
	
	for (int32_t i = 0; i < 5; i++)
	{
		std::cout << "After addition : " << h_a[i] << std::endl;
	}
	hipFree(d_a);
	hipFree(d_b);

	delete[] h_a;
	delete[] h_b;

	return EXIT_SUCCESS;
}